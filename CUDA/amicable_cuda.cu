#include <vector>
#include <fstream>
#include <iostream>
#include <string>
#include <numeric>
#include <thread>
#include <algorithm>
#include <sstream>
#include <cmath>
#include <stdexcept>
#include <cstdlib>
#include <future>
#include <functional>
#include <map>

// CUDA runtime API headers
#include <hip/hip_runtime.h>


// SIMD Intrinsics Headers (x86 specific)
#if defined(__GNUC__) || defined(__clang__)
#if defined(__x86_64__) || defined(_M_X64)
#include <immintrin.h> // x86 AVX2
#endif
#endif

//------------------------------------------------------------------------------
// Utility and Forward Declarations
//------------------------------------------------------------------------------

// A macro to check for CUDA errors.
#define CUDA_CHECK(err) \
    do { \
        hipError_t e = err; \
        if (e != hipSuccess) { \
            std::cerr << "CUDA Error: " << hipGetErrorString(e) << " at line " << __LINE__ << std::endl; \
            exit(1); \
        } \
    } while (0)

// Portability wrappers for builtins
#if defined(__GNUC__) || defined(__clang__)
#define count_trailing_zeros_u64 __builtin_ctzll
#else
// Fallback for MSVC might need _BitScanForward64
inline int count_trailing_zeros_u64(uint64_t n) {
    if (n == 0) return 64;
    int count = 0;
    while ((n & 1) == 0) {
        n >>= 1;
        count++;
        if (count >= 64) break;
    }
    return count;
}
#endif

// Global list of 32-bit primes, generated on CPU, used by GPU
static std::vector<uint32_t> primes;

// Struct to hold all data for a found pair for sorted output
struct AmicablePairOutput {
    uint64_t n;
    uint64_t s;
    std::string classification_str;
    std::string n_factors_str;
    std::string s_factors_str;
    bool operator<(const AmicablePairOutput& other) const { return n < other.n; }
};

// Struct to hold the result of amicable pair classification
struct ClassificationResult {
    std::string type_str;
};

// Function forward declarations for CPU-based logic
std::vector<std::pair<uint64_t, uint32_t>> factor(uint64_t n_val);
std::string format_factors(const std::vector<std::pair<uint64_t, uint32_t>>& factors);
ClassificationResult classify_amicable_pair(
    uint64_t n_val, uint64_t s_val,
    const std::vector<std::pair<uint64_t, uint32_t>>& n_factors,
    const std::vector<std::pair<uint64_t, uint32_t>>& s_factors);

// (CPU-based Prime Generation and Helper functions remain unchanged)
#if (defined(__GNUC__) || defined(__clang__)) && defined(__x86_64__)
__attribute__((target("avx2")))
#endif
void collect_primes_simd(const uint64_t* sieve_data, uint64_t words, uint64_t low_val_in_segment, std::vector<uint32_t>& outvec) {
    uint64_t w = 0;
#if (defined(__GNUC__) || defined(__clang__)) && defined(__AVX2__)
    for (; w + 4 <= words; w += 4) {
        __m256i v = _mm256_loadu_si256(reinterpret_cast<const __m256i*>(&sieve_data[w]));
        if (_mm256_testz_si256(v, v)) continue;
        for (int k = 0; k < 4; ++k) {
            uint64_t word = sieve_data[w + k];
            while (word) {
                int bit = count_trailing_zeros_u64(word);
                uint64_t idx_bit = (w + k) * 64 + bit;
                uint64_t prime_candidate = low_val_in_segment + 2 * idx_bit;
                if (prime_candidate <= 0xFFFFFFFFULL) outvec.push_back(static_cast<uint32_t>(prime_candidate));
                word &= word - 1;
            }
        }
    }
#endif
    for (; w < words; ++w) {
        uint64_t word = sieve_data[w];
        while (word) {
            int bit = count_trailing_zeros_u64(word);
            uint64_t idx_bit = w * 64 + bit;
            uint64_t prime_candidate = low_val_in_segment + 2 * idx_bit;
            if (prime_candidate <= 0xFFFFFFFFULL) outvec.push_back(static_cast<uint32_t>(prime_candidate));
            word &= word - 1;
        }
    }
}
std::vector<uint32_t> process_segment_task(uint32_t segment_idx_for_task, uint64_t total_sieve_slots_count_const, uint64_t segment_bits_capacity_const, const std::vector<uint32_t>& base_primes_const_ref) {
    const uint64_t current_segment_low_bit_global_idx = static_cast<uint64_t>(segment_idx_for_task) * segment_bits_capacity_const;
    if (current_segment_low_bit_global_idx >= total_sieve_slots_count_const) return {};
    const uint64_t current_segment_high_bit_global_idx = std::min(current_segment_low_bit_global_idx + segment_bits_capacity_const - 1, total_sieve_slots_count_const - 1);
    const uint64_t low_val_in_segment = 3 + 2 * current_segment_low_bit_global_idx;
    const uint64_t high_val_in_segment = 3 + 2 * current_segment_high_bit_global_idx;
    const uint64_t current_segment_active_bits = current_segment_high_bit_global_idx - current_segment_low_bit_global_idx + 1;
    const uint64_t words_in_segment = (current_segment_active_bits + 63) / 64;
    std::vector<uint64_t> sieve_local(words_in_segment);
    std::vector<uint32_t> segment_primes_local;
    sieve_local.assign(words_in_segment, ~0ULL);
    if (current_segment_active_bits % 64 != 0) sieve_local[words_in_segment - 1] &= (1ULL << (current_segment_active_bits % 64)) - 1;
    for (uint32_t p : base_primes_const_ref) {
        if (p == 2) continue;
        uint64_t p_squared = static_cast<uint64_t>(p) * p;
        if (p_squared > high_val_in_segment) break;
        uint64_t start_multiple = ((low_val_in_segment + p - 1) / p) * p;
        if (start_multiple < p_squared) start_multiple = p_squared;
        if ((start_multiple & 1) == 0) start_multiple += p;
        if (start_multiple > high_val_in_segment) continue;
        uint64_t bit_index = (start_multiple - low_val_in_segment) / 2;
        for (uint64_t b = bit_index; b < current_segment_active_bits; b += p) sieve_local[b >> 6] &= ~(1ULL << (b & 63));
    }
    double log_low_val = std::log(static_cast<double>(low_val_in_segment > 1 ? low_val_in_segment : 2.0));
    if (log_low_val < 0.1) log_low_val = 0.1;
    segment_primes_local.reserve(static_cast<size_t>(static_cast<double>(current_segment_active_bits) / log_low_val * 1.2) + 100);
    collect_primes_simd(sieve_local.data(), words_in_segment, low_val_in_segment, segment_primes_local);
    return segment_primes_local;
}
void generate_primes_multi_threaded(int mb_segment_size_param) {
    ::primes.clear();
    constexpr uint64_t MAX_SIEVE_N = 0xFFFFFFFFULL;
    const uint64_t SEGMENT_BYTES = static_cast<uint64_t>(mb_segment_size_param) * 1024 * 1024;
    const uint64_t segment_bits_capacity = SEGMENT_BYTES * 8;
    const auto sqrt_max_sieve_n = static_cast<uint32_t>(std::sqrt(static_cast<double>(MAX_SIEVE_N)));
    std::vector<bool> is_small_prime(sqrt_max_sieve_n + 1, true);
    is_small_prime[0] = is_small_prime[1] = false;
    std::vector<uint32_t> base_primes;
    double log_sqrt_n = std::log(static_cast<double>(sqrt_max_sieve_n > 1 ? sqrt_max_sieve_n : 2.0));
    if (log_sqrt_n < 0.1) log_sqrt_n = 0.1;
    base_primes.reserve(static_cast<size_t>(static_cast<double>(sqrt_max_sieve_n) / log_sqrt_n * 1.2) + 100);
    for (uint32_t i = 2; i <= sqrt_max_sieve_n; ++i) {
        if (is_small_prime[i]) {
            base_primes.push_back(i);
            for (uint64_t j = static_cast<uint64_t>(i) * i; j <= sqrt_max_sieve_n; j += i) is_small_prime[j] = false;
        }
    }
    const uint64_t total_sieve_slots_count = (MAX_SIEVE_N - 1) / 2;
    const auto num_segments = static_cast<uint32_t>((total_sieve_slots_count + segment_bits_capacity - 1) / segment_bits_capacity);
    ::primes.push_back(2);
    double log_max_n = std::log(static_cast<double>(MAX_SIEVE_N > 1 ? MAX_SIEVE_N : 2.0));
    if (log_max_n < 0.1) log_max_n = 0.1;
    ::primes.reserve(static_cast<size_t>(static_cast<double>(MAX_SIEVE_N) / log_max_n * 1.1) + 100);
    unsigned int num_hw_threads = std::thread::hardware_concurrency();
    if (num_hw_threads == 0) num_hw_threads = 1;
    num_hw_threads = std::min({num_hw_threads, 32U, num_segments > 0 ? num_segments : 1U});
    std::cout << "Generating primes in memory up to " << MAX_SIEVE_N << " (using " << mb_segment_size_param << "MB segments, " << num_hw_threads << " threads)...\n";
    std::vector<std::vector<uint32_t>> results_from_all_segments(num_segments);
    for (uint32_t batch_start_segment_idx = 0; batch_start_segment_idx < num_segments; batch_start_segment_idx += num_hw_threads) {
        std::vector<std::future<std::vector<uint32_t>>> futures_in_batch;
        futures_in_batch.reserve(num_hw_threads);
        for (unsigned int i = 0; i < num_hw_threads; ++i) {
            uint32_t current_segment_to_process = batch_start_segment_idx + i;
            if (current_segment_to_process >= num_segments) break;
            futures_in_batch.push_back(std::async(std::launch::async, process_segment_task, current_segment_to_process, total_sieve_slots_count, segment_bits_capacity, std::cref(base_primes)));
        }
        for (size_t i = 0; i < futures_in_batch.size(); ++i) {
            uint32_t actual_segment_idx_processed = batch_start_segment_idx + static_cast<uint32_t>(i);
            results_from_all_segments[actual_segment_idx_processed] = futures_in_batch[i].get();
            if (((actual_segment_idx_processed + 1) % (num_segments / 100 + 1) == 0) || (actual_segment_idx_processed == num_segments - 1)) std::cout << "\rProcessed segment " << actual_segment_idx_processed + 1 << "/" << num_segments << std::flush;
        }
    }
    std::cout << "\nAll segments processed. Consolidating primes...\n";
    for (const auto& segment_primes_vec : results_from_all_segments) ::primes.insert(::primes.end(), segment_primes_vec.begin(), segment_primes_vec.end());
    std::cout << "Prime generation complete. Found " << ::primes.size() << " primes.\n";
}

std::vector<std::pair<uint64_t, uint32_t>> factor(uint64_t n_val) {
    std::vector<std::pair<uint64_t, uint32_t>> factors_list;
    if (n_val < 2) return factors_list;
    uint64_t temp_n = n_val;
    for (uint32_t p_u32 : ::primes) {
        uint64_t p = p_u32;
        if (p * p > temp_n) break;
        if (temp_n % p == 0) {
            uint32_t exponent = 0;
            while (temp_n % p == 0) { temp_n /= p; exponent++; }
            factors_list.emplace_back(p, exponent);
        }
    }
    if (temp_n > 1) factors_list.emplace_back(temp_n, 1);
    return factors_list;
}
std::vector<std::pair<uint64_t, uint32_t>> derive_gcd_factors(const std::vector<std::pair<uint64_t, uint32_t>>& factors1, const std::vector<std::pair<uint64_t, uint32_t>>& factors2) {
    std::vector<std::pair<uint64_t, uint32_t>> gcd_factors;
    auto it1 = factors1.begin(); auto it2 = factors2.begin();
    while (it1 != factors1.end() && it2 != factors2.end()) {
        if (it1->first < it2->first) ++it1;
        else if (it2->first < it1->first) ++it2;
        else { gcd_factors.emplace_back(it1->first, std::min(it1->second, it2->second)); ++it1; ++it2; }
    }
    return gcd_factors;
}
std::vector<std::pair<uint64_t, uint32_t>> derive_quotient_factors(const std::vector<std::pair<uint64_t, uint32_t>>& num_factors, const std::vector<std::pair<uint64_t, uint32_t>>& divisor_factors) {
    std::vector<std::pair<uint64_t, uint32_t>> quotient_factors;
    auto it_num = num_factors.begin(); auto it_div = divisor_factors.begin();
    while (it_num != num_factors.end()) {
        if (it_div == divisor_factors.end() || it_num->first < it_div->first) { quotient_factors.push_back(*it_num); ++it_num; }
        else if (it_num->first > it_div->first) ++it_div;
        else { if (it_num->second > it_div->second) quotient_factors.emplace_back(it_num->first, it_num->second - it_div->second); ++it_num; ++it_div; }
    }
    return quotient_factors;
}
ClassificationResult classify_amicable_pair(uint64_t n_val, uint64_t s_val, const std::vector<std::pair<uint64_t, uint32_t>>& n_factors, const std::vector<std::pair<uint64_t, uint32_t>>& s_factors) {
    ClassificationResult result;
    uint64_t g = std::gcd(n_val, s_val);
    auto g_factors = derive_gcd_factors(n_factors, s_factors);
    auto M_factors = derive_quotient_factors(n_factors, g_factors);
    auto N_factors = derive_quotient_factors(s_factors, g_factors);
    bool sqfM = true; for (const auto& f : M_factors) if (f.second > 1) { sqfM = false; break; }
    bool sqfN = true; for (const auto& f : N_factors) if (f.second > 1) { sqfN = false; break; }
    bool cpM = (std::gcd(n_val / g, g) == 1); bool cpN = (std::gcd(s_val / g, g) == 1);
    bool regular = sqfM && sqfN && cpM && cpN;
    result.type_str = (regular ? "" : "X") + std::to_string(M_factors.size()) + "," + std::to_string(N_factors.size());
    return result;
}
std::string format_factors(const std::vector<std::pair<uint64_t, uint32_t>>& factors) {
    std::ostringstream oss;
    for (size_t i = 0; i < factors.size(); ++i) {
        auto [p, e] = factors[i]; oss << p;
        if (e > 1) oss << '^' << e;
        if (i + 1 < factors.size()) oss << '*';
    }
    return oss.str();
}

//------------------------------------------------------------------------------
// CUDA Kernel
//------------------------------------------------------------------------------

__global__ void sum_divisors_kernel(
    const unsigned long long* numbers,
    unsigned long long* results,
    const unsigned int* primes,
    const unsigned int num_primes) {

    // Calculate the globally unique thread ID
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    // Fetch the number 'n' for this thread to process
    unsigned long long n = numbers[gid];

    if (n < 2) {
        results[gid] = 0;
        return;
    }

    unsigned long long original_n = n;
    unsigned long long sum_all_divs = 1;
    unsigned long long temp_n = n;

    for (unsigned int i = 0; i < num_primes; ++i) {
        unsigned long long p = primes[i];
        if (p * p > temp_n) {
            break;
        }
        if (temp_n % p == 0) {
            unsigned long long term_sum = 1;
            unsigned long long p_power = 1;
            do {
                temp_n /= p;
                p_power *= p;
                term_sum += p_power;
            } while (temp_n % p == 0);
            sum_all_divs *= term_sum;
        }
    }

    if (temp_n > 1) {
        sum_all_divs *= (1 + temp_n);
    }

    results[gid] = sum_all_divs - original_n;
}


//------------------------------------------------------------------------------
// Main Program Logic
//------------------------------------------------------------------------------
int main(const int argc, char* argv[]) {
    // Argument parsing is unchanged
    if (argc < 2 || argc > 3) {
        std::cerr << "Usage: " << argv[0] << " <max_n (uint64)> [segment_size_mb (optional, int, default 16)]\n";
        return 1;
    }
    uint64_t max_n_arg = 0;
    try {
        max_n_arg = std::stoull(argv[1]);
    } catch (const std::exception& e) {
        std::cerr << "Error: Invalid max_n value '" << argv[1] << "'. " << e.what() << "\n";
        return 1;
    }
    if (max_n_arg < 220) {
        std::cout << "max_n (" << max_n_arg << ") is too small. Minimum is 220.\n";
        return 0;
    }
    int segment_size_mb_val = 16;
    if (argc == 3) {
        try {
            segment_size_mb_val = std::stoi(argv[2]);
            if (segment_size_mb_val < 1 || segment_size_mb_val > 2048) segment_size_mb_val = 16;
        } catch (...) { segment_size_mb_val = 16; }
    }

    // --- Phase 1: Generate Primes on CPU (unchanged) ---
    generate_primes_multi_threaded(segment_size_mb_val);
    if (::primes.empty()) {
        std::cerr << "Error: Prime generation failed.\n";
        return 1;
    }
    const uint64_t search_limit_val = max_n_arg;

    // =========================================================================
    // --- Phase 2: Amicable Pair Search on GPU (CUDA Implementation) ---
    // =========================================================================

    // 1. CUDA Device Setup
    hipDeviceProp_t deviceProp;
    int device_id = 0; // Use device 0 by default
    CUDA_CHECK(hipSetDevice(device_id));
    CUDA_CHECK(hipGetDeviceProperties(&deviceProp, device_id));
    std::cout << "Using GPU: " << deviceProp.name << std::endl;

    // 2. Prepare Data and Allocate GPU Buffers
    // OPTIMIZATION: Send only primes up to sqrt(search_limit_val) to the GPU.
    uint64_t sqrt_search_limit = static_cast<uint64_t>(std::sqrt(static_cast<double>(search_limit_val))) + 1;
    std::vector<uint32_t> gpu_primes;
    gpu_primes.reserve(static_cast<size_t>(sqrt_search_limit / std::log(sqrt_search_limit > 1 ? sqrt_search_limit : 2.0)));
    for(uint32_t p : ::primes) {
        if (p > sqrt_search_limit) break;
        gpu_primes.push_back(p);
    }
    std::cout << "Optimizing for GPU: Transferring " << gpu_primes.size() << " primes (up to sqrt(" << search_limit_val << ")) instead of " << ::primes.size() << ".\n";

    uint32_t* d_primes = nullptr;
    CUDA_CHECK(hipMalloc(&d_primes, gpu_primes.size() * sizeof(uint32_t)));
    CUDA_CHECK(hipMemcpy(d_primes, gpu_primes.data(), gpu_primes.size() * sizeof(uint32_t), hipMemcpyHostToDevice));
    uint32_t num_primes_val = static_cast<uint32_t>(gpu_primes.size());

    // --- GPU Computation in Batches ---
    std::vector<AmicablePairOutput> all_found_pairs;
    std::vector<uint64_t> s_results_cache(search_limit_val + 1);

    const size_t BATCH_SIZE_BYTES = 16 * 1024 * 1024;
    size_t batch_size_elements = BATCH_SIZE_BYTES / sizeof(uint64_t);

    std::cout << "GPU Processing in batches of " << batch_size_elements << " elements." << std::endl;

    for (uint64_t batch_start = 0; batch_start <= search_limit_val; batch_start += batch_size_elements) {
        uint64_t current_batch_end = std::min(batch_start + batch_size_elements - 1, search_limit_val);
        size_t current_batch_size = static_cast<size_t>(current_batch_end - batch_start + 1);
        if (current_batch_size == 0) continue;

        std::cout << "\rProcessing batch: " << batch_start << " to " << current_batch_end << "..." << std::flush;

        std::vector<uint64_t> numbers_to_check(current_batch_size);
        std::iota(numbers_to_check.begin(), numbers_to_check.end(), batch_start);

        unsigned long long* d_n = nullptr;
        unsigned long long* d_s = nullptr;
        CUDA_CHECK(hipMalloc(&d_n, current_batch_size * sizeof(unsigned long long)));
        CUDA_CHECK(hipMalloc(&d_s, current_batch_size * sizeof(unsigned long long)));

        CUDA_CHECK(hipMemcpy(d_n, numbers_to_check.data(), current_batch_size * sizeof(unsigned long long), hipMemcpyHostToDevice));

        // 3. Launch Kernel
        int blockSize = 256;
        int gridSize = (current_batch_size + blockSize - 1) / blockSize;
        sum_divisors_kernel<<<gridSize, blockSize>>>(d_n, d_s, d_primes, num_primes_val);
        CUDA_CHECK(hipGetLastError()); // Check for errors during kernel launch

        // 4. Copy results back to host
        CUDA_CHECK(hipMemcpy(s_results_cache.data() + batch_start, d_s, current_batch_size * sizeof(unsigned long long), hipMemcpyDeviceToHost));

        // 5. Clean up batch-specific device memory
        CUDA_CHECK(hipFree(d_n));
        CUDA_CHECK(hipFree(d_s));
    }
    // Ensure all GPU work is done before proceeding
    CUDA_CHECK(hipDeviceSynchronize());
    std::cout << "\nAll batches processed." << std::endl;

    // --- Filter candidates on CPU using the complete results cache ---
    std::cout << "Filtering potential pairs on CPU..." << std::endl;
    for (uint64_t n = 220; n <= search_limit_val; ++n) {
        if (n >= s_results_cache.size()) continue;
        uint64_t s = s_results_cache[n];
        if (s > n && s <= search_limit_val) {
             if (s < s_results_cache.size() && s_results_cache[s] == n) {
                auto nf = factor(n);
                auto sf = factor(s);
                auto cls = classify_amicable_pair(n, s, nf, sf);
                all_found_pairs.push_back({ n, s, cls.type_str, format_factors(nf), format_factors(sf) });
             }
        }
    }
    std::cout << "GPU computation and filtering complete." << std::endl;

    // --- Sort and Print Results (unchanged) ---
    std::sort(all_found_pairs.begin(), all_found_pairs.end());
    std::ofstream out_file_stream{"amicable_pairs.txt"};
    if (!out_file_stream) std::cerr << "Error: cannot open output file amicable_pairs.txt\n";
    for (const auto& p : all_found_pairs) {
        std::ostringstream oss;
        oss << p.classification_str << '\n' << p.n << '=' << p.n_factors_str << '\n' << p.s << '=' << p.s_factors_str << "\n\n";
        std::cout << oss.str();
        if(out_file_stream.is_open()) out_file_stream << oss.str();
    }
    std::cout << "Done. Found " << all_found_pairs.size() << " pairs. Results in amicable_pairs.txt\n";
    if (out_file_stream.is_open()) out_file_stream.close();

    // --- Final CUDA Cleanup ---
    CUDA_CHECK(hipFree(d_primes));

    return 0;
}